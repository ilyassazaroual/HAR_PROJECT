#include "hip/hip_runtime.h"
#include "imageProcessing.h"
#include "globalVars.h"
#include "routines.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


/*
 * blockMatchingFunction
 */


__host__ __device__ double computeMatch(unsigned char *im,
                    int im_step,
                    unsigned char *bl,
                    int bl_step,
                    int bl_cols,
                    int bl_rows,
                    int oi,
                    int oj,
                    int stride){

  if (!im || !bl) return 0.0;

  double nb = (bl_cols*bl_rows);
  double x = 0;
  for(int i = 0;i < bl_rows-stride+1;i+= stride){
  for(int j = 0;j < bl_cols-stride+1;j+= stride){
    unsigned char v1 = im[INDXs(im_step,oi+i,oj+j)];
    unsigned char v2 = bl[INDXs(bl_step,i,j)];
    x += (v2-v1)*(v2-v1);

      //im[INDXs(im_step,oi+i,oj+j)] = ABS(v2-v1);

        }
  }
  x = x / nb;
  //  printf("%f\n",x);
  return x;
}



/*
 My kernel
*/

__global__
void myKernel(double *x, int im_step, int bl_step,  unsigned char *im, unsigned char *bl, int bl_cols, int bl_rows, int stride){


    int idx =blockIdx.x * blockDim.x +threadIdx.x;
    int idy =blockIdx.y * blockDim.y +threadIdx.y;
x[im_step*idx+idy] = computeMatch(im,im_step,
                            bl,bl_step,bl_cols,bl_rows,
                            idx,idy,stride);
//printf(" je suis dans le kernel idx= %d, idy=%d la valaur du tab: %d \n----------------------------------------------------------------------------",idx,idy,x[idy+im_step*idx]);

}


double blockMatching(cv::Mat *image,
		     cv::Mat *block,
		     int stride,
		     unsigned char *res,
		     int samplenum){
  
  if (!image || !block) return DBL_MAX;
  unsigned char *bl = (unsigned char*)(block->data);
  int bl_step = block->step;
  int bl_cols = block->cols;
  int bl_rows = block->rows;

  unsigned char *im = (unsigned char*)(image->data);
  int im_step = image->step;
  int im_cols = image->cols;
  int im_rows = image->rows;

  int coord_i_min = 0;
  int coord_j_min = 0;

  double minVal =  DBL_MAX;
  int istart = 0;
  int iend =  im_rows - bl_rows;
  int jstart = 0;
  int jend =  im_cols - bl_cols;

  for(int i = istart;i < iend -stride+1;i+=stride){
    for(int j = jstart;j < jend-stride+1;j+=stride){
      double x = computeMatch(im,im_step,
			      bl,bl_step,bl_cols,bl_rows,
			      i,j,stride);
      if(x < minVal){
	minVal = x;
	coord_i_min = i;
	coord_j_min = j;
     }
    }
  }

  if (Verbose)   fprintf(stderr,"sample cols: %d\n",bl_cols);
  if (Verbose)   fprintf(stderr,"sample rows: %d\n",bl_rows);
  if (Verbose)   fprintf(stderr,"sample step: %d\n",bl_step);
  if (Verbose)   fprintf(stderr,"image cols: %d\n",im_cols);
  if (Verbose)   fprintf(stderr,"image rows: %d\n",im_rows);
  if (Verbose)   fprintf(stderr,"image step: %d\n",im_step);

  memcpy(&(res[0]),&coord_i_min,sizeof(int));
  memcpy(&(res[4]),&coord_j_min,sizeof(int));
  memcpy(&(res[8]),&minVal,sizeof(double));
  memcpy(&(res[16]),&samplenum,sizeof(int));

  if (Verbose) fprintf(stderr,"%d sample x=%d, y=%d --> %f \n",samplenum, coord_j_min,coord_i_min,minVal); 
  return minVal;
}



double blockMatchingWithScalingAndRotation(cv::Mat *image,
		     cv::Mat *blocki,
		     int stride,
		     unsigned char *res,
		     int samplenum){

  hipSetDevice(0);  
  
if (!image || !blocki) return DBL_MAX;

  //this code has memory leaks...

  unsigned char *im = (unsigned char*)(image->data);
  int im_step = image->step;
  int im_cols = image->cols;
  int im_rows = image->rows;
  
  int coord_i_min = 0;
  int coord_j_min = 0;

  double minVal =  DBL_MAX;
  float bestScale = 0;
  int bestRotation = 0;


unsigned char *cim,*cbl ;
//double res[(im_rows*im_cols)];
double *ctabX = NULL ;

 
  hipMalloc((void **)&cim, sizeof(unsigned char)*(im_rows*im_cols));
//hipMalloc((void **)&cbl, sizeof(unsigned char)*(bl_rows*bl_cols));
  hipMalloc((void **)&ctabX,sizeof(double)*(im_rows*im_cols));


  
  for (int r = -10; r < 10; r = r+2){
    printf("Trying rotation %d\n",r);
    cv::Mat *rot = rotateImage(blocki,r);
    for (float s = 1.0; s > 0.3; s = s-0.1){
      

      printf("Trying scaling %f\n",s);
      cv::Mat *block = scaleImage(rot,s);
      //showOneImage(*block);      
      
      unsigned char *bl = (unsigned char*)(block->data);
      int bl_step = block->step;
      int bl_cols = block->cols;
      int bl_rows = block->rows;      
      
      int istart = 0;
      int iend =  im_rows - bl_rows;
      int jstart = 0;
      int jend =  im_cols - bl_cols;

//unsigned char *cim,*cbl ;
double res[(im_rows*im_cols)];
//double *ctabX = NULL ;


//      for(int i = istart;i < iend -stride+1;i+=stride){
//	for(int j = jstart;j < jend-stride+1;j+=stride){
//	  double x = computeMatch(im,im_step,
//				  bl,bl_step,bl_cols,bl_rows,
//				  i,j,stride);


  dim3 blocks(jend);
  dim3 threads(1,iend);

//  hipMalloc((void **)&cim, sizeof(unsigned char)*(im_rows*im_cols));
    hipMalloc((void **)&cbl, sizeof(unsigned char)*(bl_rows*bl_cols));
//  hipMalloc((void **)&ctabX,sizeof(double)*(im_rows*im_cols));


  hipMemcpy(cim,im,sizeof(unsigned char)*(im_rows*im_cols), hipMemcpyHostToDevice );
  hipMemcpy(cbl,bl,sizeof(unsigned char)*(bl_rows*bl_cols), hipMemcpyHostToDevice );
  hipMemcpy(ctabX, res,sizeof(double)*(im_rows*im_cols), hipMemcpyHostToDevice );

  myKernel<<< blocks, threads >>>(ctabX, im_step, bl_step, cim, cbl, bl_cols,
 bl_rows, stride);

  hipMemcpy(im,cim, sizeof(unsigned char)*(im_cols*im_rows), hipMemcpyDeviceToHost );
  hipMemcpy(bl,cbl, sizeof(unsigned char)*(bl_cols*bl_rows), hipMemcpyDeviceToHost );
  hipMemcpy(res,ctabX, sizeof(double)*(im_rows*im_cols), hipMemcpyDeviceToHost ); 


for(int i = istart;i < iend -stride+1;i+=stride){
    for(int j = jstart;j < jend-stride+1;j+=stride){

//printf("pour i %d, j %d, element numero %d, valeur: %d  \n----------------------------------------------------------------------------",i,j,j+im_step*i,res[j+im_step*i]);

//printf("pour i %d, j %d, element numero %d, valeur: %d, minValeur: %d  \n-------------------------",i,j,j+im_step*i,res[j+im_step*i],minVal);
	if(res[j+im_step*i]  < minVal ){  
printf("pour i %d, j %d, element numero %d, valeur: %d, minValeur: %d  \n------------------------------------------------------",i,j,j+im_step*i,res[j+im_step*i],minVal);

	  minVal = res[j+im_step*i];
	    coord_i_min = i;
	    coord_j_min = j;
	    bestScale = s;
	    bestRotation = r;

	  }
 }
}
      
      if (Verbose)   fprintf(stderr,"sample cols: %d\n",bl_cols);
      if (Verbose)   fprintf(stderr,"sample rows: %d\n",bl_rows);
      if (Verbose)   fprintf(stderr,"sample step: %d\n",bl_step);
      if (Verbose)   fprintf(stderr,"image cols: %d\n",im_cols);
      if (Verbose)   fprintf(stderr,"image rows: %d\n",im_rows);
      if (Verbose)   fprintf(stderr,"image step: %d\n",im_step);
      if (Verbose)   fprintf(stderr,"Current score: %f\n",minVal);
      delete block; 
    }
    delete rot;
  }
  memcpy(&(res[0]),&coord_i_min,sizeof(int));
  memcpy(&(res[4]),&coord_j_min,sizeof(int));
  memcpy(&(res[8]),&minVal,sizeof(double));
  memcpy(&(res[16]),&samplenum,sizeof(int));

hipFree(cim);
hipFree(cbl);
hipFree(ctabX);

    
  if (Verbose) fprintf(stderr,"%d sample x=%d, y=%d --> %f (scale %f, rot %d) \n",
		       samplenum, coord_j_min,coord_i_min,minVal,bestScale,bestRotation); 
  return minVal;
}



